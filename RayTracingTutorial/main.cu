#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "utilities.h"

#include "color.h"
#include "hittables.h"
#include "sphere.h"
#include "camera.h"

#include "material.h"
#include "lambertian.h"
#include "metal.h"
#include "dialectric.h" 

__constant__ camera cam;

__device__ color ray_color(const ray& r, hittable** world, int depth, hiprandState* rand_state) {
    hit_record rec;

    if (depth <= 0) {
        return color(0, 0, 0);
    }

    ray curr_ray = r;
    color curr_attenuation(1, 1, 1);

    for (int i = 0; i < depth; ++i) {
        if ((*world)->hit(curr_ray, 0.001, infinity, rec)) {
            ray scatter;
            color attenuation;
            if (rec.mat_ptr->scatter(curr_ray, rec, attenuation, scatter, rand_state)) {
                curr_ray = scatter;
                curr_attenuation *= attenuation;
            } else {
                return color(0, 0, 0);
            }
        } else {
            vec3 unit_dir = unit_vector(curr_ray.direction());
            double t = 0.5 * (unit_dir.y() + 1.0);
            return curr_attenuation * ((1.0 - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0));
        }
    }
    return color(0.0, 0.0, 0.0);
}

__global__ void rand_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1993, 0, 0, rand_state);
    }
}

__global__ void free_world(hittable** d_objects, int num_hittables, hittable** d_world) {
    for (int i = 0; i < num_hittables; ++i) {
        delete ((sphere*)d_objects[i])->get_mat_ptr();
        delete d_objects[i];
    }
    delete *d_world;
}

__global__ void random_scene(hittable** d_objects, hittable** d_world, hiprandState* rand_state) {
    if (threadIdx.x != 0 || blockIdx.x != 0) {
        return;
    }

    hiprandState local_rand_state = *rand_state;
    d_objects[0] = new sphere(point3(0, -1000, 0), 1000, new lambertian(color(0.5, 0.5, 0.5)));

    int i = 1;

    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            auto choose_mat = random_double(&local_rand_state);
            point3 center(a + 0.9 * random_double(&local_rand_state), 0.2, b + 0.9 * random_double(&local_rand_state));

            if ((center - point3(4, 0.2, 0)).length() > 0.9) {

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = color::random(&local_rand_state) * color::random(&local_rand_state);
                    d_objects[i++] = new sphere(center, 0.2, new lambertian(albedo));
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = (color::random(&local_rand_state) + 1.0) * 0.5;
                    auto fuzz = random_double(&local_rand_state) * 0.5;
                    d_objects[i++] = new sphere(center, 0.2, new metal(albedo, fuzz));
                } else {
                    // glass
                    d_objects[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
    }

    d_objects[i++] = new sphere(point3(0, 1, 0), 1.0, new dielectric(1.5));
    d_objects[i++] = new sphere(point3(-4, 1, 0), 1.0, new lambertian(color(0.4, 0.2, 0.1)));
    d_objects[i++] = new sphere(point3(4, 1, 0), 1.0, new metal(color(0.7, 0.6, 0.5), 0.0));

    *rand_state = local_rand_state;
    *d_world = (hittable*) new hittables(d_objects, i);
}

__global__ void render_init(int image_width, int image_height, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= image_width || j >= image_height) {
        return;
    }

    int pixel_index = j * image_width + i;
    hiprand_init(1993, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(color* image_buffer, int image_width, int image_height, int samples_per_pixel, int max_depth, hittable** world, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= image_width || j >= image_height) {
        return;
    }

    int pixel_index = j * image_width + i;
    hiprandState local_rand_state = rand_state[pixel_index];

    color pixel_color(0, 0, 0);
    for (int sample = 0; sample < samples_per_pixel; ++sample) {
        double u = (i * 1.0 + random_double(&local_rand_state)) / image_width;
        double v = (j * 1.0 + random_double(&local_rand_state)) / image_height;
        ray r = cam.get_ray(u, v, &local_rand_state);
        pixel_color += ray_color(r, world, max_depth, &local_rand_state);
    }

    image_buffer[pixel_index] = pixel_color;
    rand_state[pixel_index] = local_rand_state;
}

void init_host_image_buffer(color* image_buffer, int image_width, int image_height) {
    for (int row = 0; row < image_height; ++row) {
        for (int col = 0; col < image_width; ++col) {
            image_buffer[row * image_width + col] = color(0, 0, 0);
        }
    }
}

int main() {
    // initialize render config
    auto aspect_ratio = 3.0 / 2.0;
    auto image_width = 200;         // 1200
    auto image_height = static_cast<int>(image_width / aspect_ratio);
    auto samples_per_pixel = 10;    // 500
    auto max_depth = 50;

    // initialize camera
    point3 lookfrom(13, 2, 3);
    point3 lookat(0, 0, 0);
    vec3 vup(0, 1, 0);
    auto dist_to_focus = 10.0;
    double aperture = 0.1;
    camera host_cam(lookfrom, lookat, vup, 20.0, aspect_ratio, aperture, dist_to_focus);

    std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    unsigned int size_image_buffer = image_width * image_height;
    unsigned int mem_size_image_buffer = size_image_buffer * sizeof(color);
    color* image_buffer, * d_image_buffer;
    hipStream_t stream_image_buffer, stream_camera, stream_world;

    checkCudaErrors(hipStreamCreate(&stream_image_buffer));
    checkCudaErrors(hipStreamCreate(&stream_camera));
    checkCudaErrors(hipStreamCreate(&stream_world));

    // copy camera data over to device memory
    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(cam), &host_cam, sizeof(camera), 0, hipMemcpyDefault, stream_camera));

    // initialize world
    hiprandState* d_rand_state_create_world;
    checkCudaErrors(hipMalloc(&d_rand_state_create_world, sizeof(hiprandState)));

    rand_init <<<1, 1, 0, stream_world>>> (d_rand_state_create_world);
    checkCudaErrors(hipGetLastError());

    hittable** d_objects;
    hittable** d_world;
    int num_hittables = 485;
    checkCudaErrors(hipMalloc(&d_objects, num_hittables * sizeof(hittable*)));
    checkCudaErrors(hipMalloc(&d_world, sizeof(hittable*)));

    random_scene <<<1,1,0,stream_world>>>(d_objects, d_world, d_rand_state_create_world);

    // copy image data buffer to device memory
    std::cerr << "Image width: " << image_width << " image height: " << image_height << "\n";
    std::cerr << "Allocating " << size_image_buffer << " number of pixels with " << mem_size_image_buffer << " bytes on host and device.\n";

    checkCudaErrors(hipHostMalloc(&image_buffer, mem_size_image_buffer));
    init_host_image_buffer(image_buffer, image_width, image_height);

    checkCudaErrors(hipMalloc(&d_image_buffer, mem_size_image_buffer));
    checkCudaErrors(hipMemcpyAsync(d_image_buffer, image_buffer, mem_size_image_buffer, hipMemcpyHostToDevice, stream_image_buffer));

    // wait for render initialization to finish
    checkCudaErrors(hipStreamSynchronize(stream_image_buffer));
    checkCudaErrors(hipStreamSynchronize(stream_camera));
    checkCudaErrors(hipStreamSynchronize(stream_world));

    // prepare rendering with a hiprand state per pixel
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid((image_width + threads_per_block.x - 1) / threads_per_block.x, (image_height + threads_per_block.y - 1) / threads_per_block.y);

    hiprandState* d_rand_state_render;
    checkCudaErrors(hipMalloc(&d_rand_state_render, image_width * image_height * sizeof(hiprandState)));

    render_init <<<blocks_per_grid, threads_per_block, 0, stream_image_buffer>>> (image_width, image_height, d_rand_state_render);
    checkCudaErrors(hipGetLastError());

    render <<<blocks_per_grid, threads_per_block, 0, stream_image_buffer>>> (d_image_buffer, image_width, image_height, samples_per_pixel, max_depth, d_world, d_rand_state_render);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpyAsync(image_buffer, d_image_buffer, mem_size_image_buffer, hipMemcpyDeviceToHost, stream_image_buffer));
    checkCudaErrors(hipStreamSynchronize(stream_image_buffer));

    std::cerr << "Writing result from device to host\n";

    for (int row = image_height - 1; row >= 0; --row) {
        for (int col = 0; col < image_width; ++col) {
            write_color(std::cout, image_buffer[row * image_width + col], samples_per_pixel);
        }
    }

    std::cerr << "\nDone.\n";

    free_world <<<1, 1, 0, stream_world>>> (d_objects, num_hittables, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipStreamSynchronize(stream_world));

    checkCudaErrors(hipStreamDestroy(stream_world));
    checkCudaErrors(hipStreamDestroy(stream_camera));
    checkCudaErrors(hipStreamDestroy(stream_image_buffer));

    checkCudaErrors(hipHostFree(image_buffer));
    checkCudaErrors(hipFree(d_image_buffer));

    checkCudaErrors(hipFree(d_rand_state_create_world));
    checkCudaErrors(hipFree(d_rand_state_render));
    checkCudaErrors(hipFree(d_objects));
    checkCudaErrors(hipFree(d_world));
}